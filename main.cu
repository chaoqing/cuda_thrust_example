#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

using namespace std;
using namespace boost::numeric::odeint;

/****************  使用的数据结构定义  ********************/
typedef float ElemType;//数据类型

typedef thrust::device_vector<ElemType> StateVector;//系统状态
typedef StateVector::iterator StateVectorIterator;//迭代器
typedef thrust::host_vector<ElemType> HStateVector;
typedef HStateVector::iterator HStateVectorIterator;

typedef StateVector ElemTypeVector;

typedef unsigned int Index;
typedef thrust::device_vector<Index> IndexVector;

/****************  一些系统常量定义  ********************/
const Index DIM=2;//系统维数
const Index N0=101;//系统大小
const Index REP=201;//系综大小
const Index N=N0*REP;//系统总大小
const Index LENGTH=N*DIM;//状态总长
const ElemType PI = 3.1415926535897932384626433832795029;
/****************  方程参数常量  ********************/
const ElemType mu=1.00;//分岔参数
const ElemType dr=0.15;//耦合参数实部
const ElemType di=-1.0;//耦合参数虚部
const ElemType b=1.0;//阻尼项
/****************  方程参数区间常量  ********************/
const ElemType omega_from=0.95*2*PI;
const ElemType omega_to=1.05*2*PI;
const ElemType couple_k_from=0.0;
const ElemType couple_k_to=10.0;
/****************  系统运行参数  ********************/
const ElemType dt = 1.0/360.0/1.0;//时间步长
const ElemType t_transients = 100.0;//暂态时长
const ElemType t_max = 300.0;//总时长

/****************  需要用到的(仿)函数  ********************/
struct plus_r//计算序参量
{
    template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )
        {
            thrust::get<0>(t) += sqrt(thrust::get<1>(t)*thrust::get<1>(t)+thrust::get<2>(t)*thrust::get<2>(t));
        }
};

IndexVector rep_each(Index n, Index each)//[0,0,0, 1,1,1, 2,2,2, 3,3,3 ...] each=3
{
    IndexVector x(n);
    thrust::counting_iterator<Index> first(0);//计数迭代器
    thrust::transform(first, first+n,
            thrust::make_constant_iterator(each),//常量
            x.begin(),
            thrust::divides<Index>());
    return x;
}

IndexVector rep_times(Index n, Index len)//[0,1,2,...len-1, 0,1,2,...len-1, 0,1,2,...len-1]
{
    IndexVector x(n);
    thrust::counting_iterator<Index> first(0);//计数迭代器
    thrust::transform(first, first+n,
            thrust::make_constant_iterator(len),//常量
            x.begin(),
            thrust::modulus<Index>());
    return x;
}


void mean_field(//平均场
        ElemTypeVector &xmean, ElemTypeVector &ymean,//存放结果
        const StateVector &x, const IndexVector &group)//系统状态及系综分组
{
    static IndexVector useless(REP);//寄存系综序号集合,这个保存下来也没意思
    thrust::reduce_by_key(
            group.begin() ,group.end(),
            x.begin(), useless.begin(), xmean.begin(),
            thrust::equal_to<Index>(),
            thrust::plus<ElemType>());
    thrust::reduce_by_key(
            group.begin(),group.end(),
            x.begin()+N, useless.begin(), ymean.begin(),
            thrust::equal_to<Index>(),
            thrust::plus<ElemType>());

    thrust::transform(xmean.begin(), xmean.end(),
            thrust::make_constant_iterator((ElemType)N0),//常量
            xmean.begin(),
            thrust::divides<ElemType>());
    thrust::transform(ymean.begin(), ymean.end(),
            thrust::make_constant_iterator((ElemType)N0),//常量
            ymean.begin(),
            thrust::divides<ElemType>());
}

struct SYSTEM
{
    /****************  参数  ********************/
    ElemTypeVector k;//耦合强度
    ElemTypeVector omega;//自身频率
    ElemType o;//驱动频率
    ElemType f;//驱动强度

    /****************  一些需要用到的临时变量  ********************/
    ElemTypeVector xmean;//x坐标平均值
    ElemTypeVector ymean;//y坐标平均值
    IndexVector by_N0;
    IndexVector by_REP;

    /****************  设置参数  ********************/
    void set_omega(ElemType from=0.95*2*PI, ElemType to=1.05*2*PI){
        thrust::sequence(omega.begin(), omega.end(), from, (to-from)/(N0-1));
    }

    void set_k(ElemType from=0.0, ElemType to=1.0){
        thrust::sequence(k.begin(), k.end(), from, (to-from)/(REP-1));
    }


    SYSTEM(){
        omega.resize(N0);//各个子系统对应位置个体omega有相同值
        k.resize(REP);//各个子系统内部使用统一的耦合强度

        xmean.resize(REP);//记录各个子系统平均场
        ymean.resize(REP);
        by_N0=rep_times(N, N0);//按N0划分[0,1,2...N0-1,0,1,2...N0-1, ...... 0,1,2...N0-1] times=REP
        by_REP=rep_each(N, N0);//按系综划分[0,0,0,... 1,1,1,... 2,2,2..., ......,  REP-1,REP-1,REP-1,...] REP=N/N0,each=N0
    }

    struct function//单个振子的方程
    {
        ElemType dt;
        function(ElemType _dt):dt(_dt){}
        template< class Tuple >
            __host__ __device__
            void operator()( Tuple t )
            {
                /****************  定义一些宏来简化变量提取的过程  ********************/
#define value(x,index) thrust::get<index>((x))

#define x value(value(t,0),0)
#define y value(value(t,0),1)

#define xmean value(value(t,1),0)
#define ymean value(value(t,1),1)
#define k value(value(t,1),2)
#define omega value(value(t,1),3)
#define o value(value(t,1),4)
#define f value(value(t,1),5)

#define dxdt value(value(t,2),0)
#define dydt value(value(t,2),1)

                //ElemType x=value(value(t,0),0);
                //ElemType y=value(value(t,0),1);
                ElemType mo=x*x+y*y;

                dxdt = mu*x-omega*y + k*(dr*(xmean-x)-di*(ymean-y)) - b*mo*x+ f*cos(o*dt);
                dydt = mu*y+omega*x + k*(dr*(ymean-y)+di*(xmean-x)) - b*mo*y+ f*sin(o*dt);
#undef x
#undef y

#undef xmean
#undef ymean
#undef k
#undef omega
#undef o
#undef f

#undef dxdt
#undef dydt

#undef value
            }
    };

    void operator() ( const StateVector &x , StateVector &dxdt , const ElemType dt )
    {
        /****************  首先定义几个常用的操作  ********************/
#define ppi(x,y) /*permutation_parameter_iterator*/\
        thrust::make_permutation_iterator((x),(y)) //创建一个排列迭代器
#define Nspi(x) /*N_scale_parameter_iterator*/\
        thrust::make_constant_iterator((x)) //创建一个常量迭代器，在所有系综中通用参数
#define szi(x) /*state_zip_iterator*/\
        thrust::make_zip_iterator(thrust::make_tuple((x),(x)+N))//将各维分量拆开后打包
#define pzi(k,o,f,omega,xm,ym) /*parameter_zip_iterator*/\
        thrust::make_zip_iterator(thrust::make_tuple((k),(o),(f),(omega),(xm),(ym)))//将参数打包

        mean_field(xmean, ymean, x, by_REP);//更新平均场
        thrust::for_each(
                thrust::make_zip_iterator(//begin
                    thrust::make_tuple(
                        szi(x.begin()),//x
                        pzi(ppi(xmean.begin(),by_REP.begin()),//xmean
                            ppi(ymean.begin(),by_REP.begin()),//ymean
                            ppi(k.begin(), by_REP.begin()),//k
                            ppi(omega.begin(),by_N0.begin()),//omega
                            Nspi(o),//force omega
                            Nspi(f)//force strength
                           ),
                        szi(dxdt.begin()))),//dxdt 

                thrust::make_zip_iterator(//end
                    thrust::make_tuple(
                        szi(x.begin()+N),//x
                        pzi(ppi(xmean.begin(),by_REP.end()),//xmean
                            ppi(ymean.begin(),by_REP.end()),//ymean
                            ppi(k.begin(), by_REP.end()),//k
                            ppi(omega.begin(),by_N0.end()),//omega
                            Nspi(o),//force omega
                            Nspi(f)//force strength
                           ),
                        szi(dxdt.begin()+N))),//dxdt 

                function(dt)
                    );
#undef ppi
#undef Nspi
#undef szi
#undef pzi
    }
};


struct observer
{
    ElemTypeVector m_r;
    Index m_count;
    /****************  一些需要用到的临时变量  ********************/
    ElemTypeVector xmean;//x坐标平均值
    ElemTypeVector ymean;//y坐标平均值
    IndexVector by_N0;
    IndexVector by_REP;

    observer():m_count(0) {
        m_r.resize(REP);
        thrust::fill(m_r.begin(),m_r.end(),(ElemType)0.0);
        xmean.resize(REP);//记录各个子系统平均场
        ymean.resize(REP);
        by_N0=rep_times(N, N0);//按N0划分[0,1,2...N0-1,0,1,2...N0-1, ...... 0,1,2...N0-1] times=REP
        by_REP=rep_each(N, N0);//按系综划分[0,0,0,... 1,1,1,... 2,2,2..., ......,  REP-1,REP-1,REP-1,...] REP=N/N0,each=N0
    }

    template< class State >
        void operator()( const State &x , ElemType t )
        {
            mean_field(xmean, ymean, x, by_REP);//更新平均场
            thrust::for_each(
                    thrust::make_zip_iterator(//begin
                        thrust::make_tuple(
                            m_r.begin(),
                            xmean.begin(),
                            ymean.begin())), 

                    thrust::make_zip_iterator(//end
                        thrust::make_tuple(
                            m_r.end(),
                            xmean.end(),
                            ymean.end())), 
                    plus_r());
            ++m_count;
        }

    void report() {
        if(m_count!=0){
            thrust::transform(m_r.begin(), m_r.end(),
                    thrust::make_constant_iterator((ElemType)m_count),//常量
                    m_r.begin(),
                    thrust::divides<ElemType>());
        }
        thrust::copy(m_r.begin(),m_r.end(),std::ostream_iterator<ElemType>(std::cout,"\n"));
    }

    void reset( void ) {
        thrust::fill(m_r.begin(),m_r.end(),(ElemType)0.0);
        m_count = 0; 
    }
};


/****************  主函数开始  ********************/
int main( int arc , char* argv[] )
{
    HStateVector xinit_h( LENGTH );
    for( Index i=0 ; i<LENGTH ; ++i )
    {
        if(i<LENGTH/2&&i%N0==0)//x且一个新的子系统
        {
            xinit_h[i] = 0.1;
        }
        else{
            xinit_h[i] = 0.0;
        }
        //xinit_h[i] = drand48();
    }


    SYSTEM sys;
    sys.f=0.0;
    sys.o=0.0;

    typedef runge_kutta4< StateVector , ElemType , StateVector , ElemType > stepper_type;

    sys.set_k(couple_k_from,couple_k_to);
    sys.set_omega(omega_from,omega_to);
    
    observer obs;

    StateVector xinit = xinit_h;

    Index steps1 = integrate_const( stepper_type() , boost::ref( sys ) , xinit , (ElemType)0.0 , t_transients , dt );
    Index steps2 = integrate_const( stepper_type() , boost::ref( sys ) , xinit , (ElemType)0.0 , t_max , dt , boost::ref( obs ) );
    obs.report();

    return 0;
}
